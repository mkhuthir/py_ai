
#include <hip/hip_runtime.h>
#define N 10000000

__global__ 
void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory (CPU)
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Allocate device memory (GPU)
    hipMalloc((void**)&d_a  , sizeof(float) * N);
    hipMalloc((void**)&d_b  , sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Initialize array (CPU)
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Transfer data from host to device memory (CPU>GPU)
    hipMemcpy(d_a  , a,   sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b  , b,   sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);
}
